
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <omp.h>
#include <math.h>       /* fabsf */
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#define DEBUG 0

//Error check-----
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
//Error check-----
//This is a very good idea to wrap your calls with that function.. Otherwise you will not be able to see what is the error.
//Moreover, you may also want to look at how to use cuda-memcheck and cuda-gdb for debugging.


__global__ void find_result_from_matrix(int *adj, int *xadj, int *values, int size, int *cycle_result, int cycle_length)
{
	int step = 32*1024;
	int id = blockDim.x * blockIdx.x + threadIdx.x; //Global id for the thread
	for(int i = id; i<size; i+= step)
	{
		cycle_result[i] = 0;
		for(int col_iter = xadj[i]; col_iter < xadj[i+1]; col_iter++)
		{
			int col_ptr = adj[col_iter];
			if(col_ptr == i)
			{
				cycle_result[i] = values[col_iter]/(2*cycle_length);
			}
		}
	}
}


__global__ void multiply_matrix(int *adj, int *xadj, int *tadj, int *txadj,int* values, int *tvalues, int size, int * lookup_place, int *res_adj, int *res_values)
{
	int step = 32 * 1024;
	int id = blockDim.x * blockIdx.x + threadIdx.x; //Global id for the thread
	for(int i = id; i<size; i+= step)
	{
		int result_size = 0;
		//For each non zero element 
		for(int col_iter = xadj[i]; col_iter < xadj[i+1]; col_iter++)
		{
			int col_ptr = adj[col_iter];
			int place = 0;
			//for each non zero element
			for(int row_iter = txadj[col_ptr]; row_iter < txadj[col_ptr+1]; row_iter++)
			{
				int row_ptr = tadj[row_iter];
				int value = values[col_iter] * tvalues[row_iter];
				int place_to_enter_data = lookup_place[i]+place;
				if(result_size <= place)
				{
					res_adj[place_to_enter_data] = row_ptr;
					res_values[place_to_enter_data] = value;
					result_size++;
				}else
				{
					res_values[place_to_enter_data] += value;
				}
				place++;
			}
		}
	}
}

__global__ void find_required_memory(int *adj, int *xadj, int *tadj, int *txadj, int size, int* result)
{
	int step = 32 * 1024;
	int id = blockDim.x * blockIdx.x + threadIdx.x; //Global id for the thread
	for(int i = id; i<size; i+= step)
	{
		int result_size = 0;
		//For each non zero element 
		for(int col_iter = xadj[i]; col_iter < xadj[i+1]; col_iter++)
		{
			int col_ptr = adj[col_iter];
			int place = 0;
			//for each non zero element
			for(int row_iter = txadj[col_ptr]; row_iter < txadj[col_ptr+1]; row_iter++)
			{
				if(result_size <= place)
				{
					result_size++;
				}
				place++;
			}
		}
		result[i] += result_size;
	}
}



void wrapper(int* adj, int* xadj, int* tadj, int* txadj, int* values, int *tvalues, int* &res_adj,int *&res_xadj, int  *&res_values, int size, int adj_size, int xadj_size, int tadj_size, int txadj_size, int values_size, int tvalues_size, int &result_adj_size, int &result_xadj_size, int &result_values_size){
  
  int* d_adj, *d_xadj, *d_tadj, *d_txadj, *d_values, *d_tvalues, *d_res_adj, *d_res_values,*d_res_size;

  int *res_size = (int*)malloc(size * sizeof(int));
  memset(res_size, 0 ,(size)*sizeof(int));

  hipMalloc(&d_adj, adj_size * sizeof(int));
  hipMalloc(&d_xadj, xadj_size * sizeof(int));
  hipMalloc(&d_tadj, tadj_size * sizeof(int));
  hipMalloc(&d_txadj, txadj_size * sizeof(int));
  hipMalloc(&d_values, values_size * sizeof(int));
  hipMalloc(&d_tvalues, tvalues_size * sizeof(int));
  hipMalloc(&d_res_size, size * sizeof(int));

  hipMemcpy(d_res_size, res_size, (size) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_adj,adj,adj_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_xadj,xadj,xadj_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_tadj,tadj,tadj_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_txadj,txadj,txadj_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_values,values,values_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_tvalues,tvalues,tvalues_size*sizeof(int), hipMemcpyHostToDevice);
  
  find_required_memory<<<32,1024>>>(d_adj, d_xadj, d_tadj, d_txadj,size,d_res_size);
  gpuErrchk( hipDeviceSynchronize() );
  hipMemcpy(res_size, d_res_size, size * sizeof(int), hipMemcpyDeviceToHost);
  int sum = 0;
  #pragma omp parallel for reduction(+: sum)
  for(int i = 0; i< size; i++)
  {
	sum+=res_size[i];
  }
  hipMalloc(&d_res_values, (sum) * sizeof(int));
  hipMalloc(&d_res_adj, (sum) * sizeof(int));

  //TODO(kaya) : find another solution for this = parallel.
  int *place_look_up = (int*)malloc(size * sizeof(int));
  int *d_place_look_up;
  hipMalloc(&d_place_look_up, size*sizeof(int));
  place_look_up[0] = 0; 
  for(int i = 1; i< size; i++)
  {
	place_look_up[i] = place_look_up[i-1] + res_size[i-1];
  } 
  hipMemcpy(d_place_look_up, place_look_up, size*sizeof(int), hipMemcpyHostToDevice);
  //place_look_up will be used to determine where to enter the adj and values data for each line
  multiply_matrix<<<32,1024>>>(d_adj,d_xadj,d_tadj,d_txadj,d_values,d_tvalues,size,d_place_look_up,d_res_adj,d_res_values);
  gpuErrchk( hipDeviceSynchronize() );
  res_adj = (int*)(malloc(sum * sizeof(int)));
  res_values = (int*)(malloc(sum * sizeof(int)));
  hipMemcpy(res_adj,d_res_adj, sum*sizeof(int), hipMemcpyDeviceToHost);  
  hipMemcpy(res_values,d_res_values, sum*sizeof(int), hipMemcpyDeviceToHost);  
  res_xadj = (int*)malloc((size+1) * sizeof(int));
  for(int i = 0; i<size; i++)
  {
	res_xadj[i] = place_look_up[i];
  } 
  res_xadj[size] = sum;
  result_adj_size = sum;
  result_xadj_size = size+1;
  result_values_size = sum;
  printf("multiplication done\n");
  free(place_look_up);
  free(res_size);
  hipFree(d_adj);
  hipFree(d_xadj);
  hipFree(d_tadj);
  hipFree(d_txadj);
  hipFree(d_values);
  hipFree(d_tvalues);
  hipFree(d_res_size);
  hipFree(d_place_look_up);
}
void find_result_gpu(std::vector<int> &adj, std::vector<int> &xadj, std::vector<int> &values, int size, int cycle_length)
{
  	float elapsedTime;
  	hipEvent_t start,stop; 
  	hipEventCreate(&start);
  	hipEventRecord(start, 0);
	int *adj_ptr = &adj[0];
	int *xadj_ptr = &xadj[0];
	int *values_ptr = &values[0];

	int *result_adj;
	int  result_adj_size = 0;
	int *result_xadj;
	int  result_xadj_size = 0;
	int *result_values;
	int  result_values_size = 0;
	if(cycle_length == 3)
	{
		int *pa_result_adj;
		int  pa_result_adj_size = 0;
		int *pa_result_xadj;
		int  pa_result_xadj_size = 0;
		int *pa_result_values;
		int  pa_result_values_size = 0;
		wrapper(adj_ptr, xadj_ptr, adj_ptr, xadj_ptr, values_ptr, values_ptr,pa_result_adj,pa_result_xadj,pa_result_values,size,adj.size(), xadj.size(),adj.size(),xadj.size(), values.size(), values.size(),pa_result_adj_size,pa_result_xadj_size,pa_result_values_size);
		wrapper(pa_result_adj, pa_result_xadj, adj_ptr, xadj_ptr, pa_result_values, values_ptr,result_adj,result_xadj,result_values,size,pa_result_adj_size, pa_result_xadj_size,adj.size(),xadj.size(), pa_result_values_size, values.size(),result_adj_size,result_xadj_size,result_values_size);
		//A*A*A => R
		
		free(pa_result_adj);
		free(pa_result_xadj);
		free(pa_result_values);
	}else if(cycle_length == 4)
	{
		int *pa_result_adj;
		int  pa_result_adj_size = 0;
		int *pa_result_xadj;
		int  pa_result_xadj_size = 0;
		int *pa_result_values;
		int  pa_result_values_size = 0;
		wrapper(adj_ptr, xadj_ptr, adj_ptr, xadj_ptr, values_ptr, values_ptr,pa_result_adj,pa_result_xadj,pa_result_values,size,adj.size(), xadj.size(),adj.size(),xadj.size(), values.size(), values.size(),pa_result_adj_size,pa_result_xadj_size,pa_result_values_size);
		wrapper(pa_result_adj, pa_result_xadj, pa_result_adj, pa_result_xadj, pa_result_values, pa_result_values,result_adj,result_xadj,result_values,size,pa_result_adj_size, pa_result_xadj_size,pa_result_adj_size,pa_result_xadj_size, pa_result_values_size, pa_result_values_size,result_adj_size,result_xadj_size,result_values_size);

		free(pa_result_adj);
		free(pa_result_xadj);
		free(pa_result_values);
		//A*A => A2 * A2 => R
	}else if(cycle_length == 5)
	{
		int *pa_result_adj;
		int  pa_result_adj_size = 0;
		int *pa_result_xadj;
		int  pa_result_xadj_size = 0;
		int *pa_result_values;
		int  pa_result_values_size = 0;
		int *pa_result2_adj;
		int  pa_result2_adj_size = 0;
		int *pa_result2_xadj;
		int  pa_result2_xadj_size = 0;
		int *pa_result2_values;
		int  pa_result2_values_size = 0;
		wrapper(adj_ptr, xadj_ptr, adj_ptr, xadj_ptr, values_ptr, values_ptr,pa_result_adj,pa_result_xadj,pa_result_values,size,adj.size(), xadj.size(),adj.size(),xadj.size(), values.size(), values.size(),pa_result_adj_size,pa_result_xadj_size,pa_result_values_size);
		wrapper(pa_result_adj, pa_result_xadj, pa_result_adj, pa_result_xadj, pa_result_values, pa_result_values,pa_result2_adj,pa_result2_xadj,pa_result2_values,size,pa_result_adj_size, pa_result_xadj_size,pa_result_adj_size,pa_result_xadj_size, pa_result_values_size, pa_result_values_size,pa_result2_adj_size,pa_result2_xadj_size,pa_result2_values_size);
		wrapper(pa_result2_adj, pa_result2_xadj, adj_ptr, xadj_ptr, pa_result2_values, values_ptr,result_adj,result_xadj,result_values,size,pa_result2_adj_size, pa_result2_xadj_size,adj.size(),xadj.size(), pa_result2_values_size, values.size(),result_adj_size,result_xadj_size,result_values_size);

		//TODO(kaya): find the cycle result
		free(pa_result_adj);
		free(pa_result_xadj);
		free(pa_result_values);
		free(pa_result2_adj);
		free(pa_result2_xadj);
		free(pa_result2_values);
		//A*A => A2 * A2 * A => R
	}
	int *d_adj, *d_xadj, *d_values, *d_cycle_result;
	hipMalloc(&d_adj, result_adj_size * sizeof(int));
	hipMalloc(&d_xadj, result_xadj_size * sizeof(int));
	hipMalloc(&d_values, result_values_size * sizeof(int));
	hipMalloc(&d_cycle_result, size * sizeof(int));
	hipMemcpy(d_adj, result_adj, result_adj_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_xadj, result_xadj, result_xadj_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_values, result_values, result_values_size * sizeof(int), hipMemcpyHostToDevice);
	find_result_from_matrix<<<32,1024>>>(d_adj, d_xadj, d_values, size, d_cycle_result,cycle_length);	
  	gpuErrchk( hipDeviceSynchronize() );
	int* result_cycle = (int*)malloc(size * sizeof(int));	
	hipMemcpy(result_cycle, d_cycle_result, size * sizeof(int), hipMemcpyDeviceToHost);
  	
	hipEventCreate(&stop);
  	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop);
  	
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Elapsed time: %f\n",elapsedTime);
	/*
	for(int i = 0; i<size; i++)
	{
		printf("%d	%d\n", i, result_cycle[i]);
	}
	*/
	hipFree(d_adj);
	hipFree(d_xadj);
	hipFree(d_values);
	hipFree(d_cycle_result);
	free(result_adj);
	free(result_xadj);
	free(result_values);
	free(result_cycle);
}
